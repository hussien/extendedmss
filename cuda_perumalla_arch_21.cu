#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <thrust/scan.h>
#include <climits>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <omp.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/host_vector.h>
#include <numeric>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#define THREAD_NUM 1024
#define BLOCKS_SCHED 2
#define SIZE_WARP 32

/*Definição da operação de máximo, usado nas operações de prefixo e sufixo máximo*/
thrust::maximum<int> max_op;
thrust::minimum<int> min_op;
thrust::plus<int> plus_op;

thrust::bit_and<int> bit_and_op;

/*for use _1 function in thrust::for_each function*/
using namespace thrust::placeholders;

typedef thrust::device_vector<int>::iterator Iterator; 

struct calcM : public thrust::unary_function<thrust::tuple<int,int,int,int,int>,int>
{
    	__host__ __device__
        float operator()(const thrust::tuple<int,int,int,int,int>& a) const
        {
		//0 = d_q, 1 = d_psum, 2 = d_ssum, 3 = d_pmax, 4 = d_smax
		//m[i] = (pmax[i] - ss[i] + q[i]) + (smax[i] - ps[i] + q[i]) - q[i];		
		return (thrust::get<3>(a) - thrust::get<2>(a) + thrust::get<0>(a)) +
		       (thrust::get<4>(a) - thrust::get<1>(a) + thrust::get<0>(a)) -
		       thrust::get<0>(a);
        }
};


/*for transform: m[i] == global_max ? -1 : 0;*/
struct binaryTransform
{
	const int constant;

	binaryTransform(int _constant) : constant(_constant) {}

	__host__ __device__
	void operator()( int& e) const 
	{
		e = e == constant ? -1 : 0;
	}
};

__global__ void findRelatedSolutions(int * v, int partition, int dev_id, int boundary, int * device_max, int * device_min, int * device_total) {
	int tid = threadIdx.x  + blockIdx.x * blockDim.x;
	int total_p = blockDim.x * gridDim.x;

	unsigned short int idWarp = threadIdx.x >> 5;
	unsigned short int idtW = threadIdx.x % SIZE_WARP;
	int valueNeighbor;

	int myvalue = 0;
	int next = 0;

	int mymax = INT_MIN;
	int mymin = INT_MAX;
	int mytotal = 0;

	extern __shared__ int array[];
	int* block_max = (int*)&array[0*blockDim.x];
	int* block_min = (int*)&array[1*blockDim.x];
	int* block_total = (int*)&array[2*blockDim.x];

	block_max[threadIdx.x] = INT_MIN;
	block_min[threadIdx.x] = INT_MAX;
	block_total[threadIdx.x] = 0;

	__syncthreads();

	#pragma unroll 128
	for(int i = tid; i < partition; i += total_p) {
		myvalue = v[i];
		next = i < partition - 1 ? v[i+1] : 0;

		if(myvalue > 0 && next == 0) {
			if((i != partition - 1) || (i == partition - 1 && boundary == 0)) {
				mytotal++;
				mymax = myvalue > mymax ? myvalue : mymax;
				mymin = myvalue < mymin ? myvalue : mymin;
			}
		}
	}

	block_max[threadIdx.x] 	 = mymax;
	block_min[threadIdx.x]	 = mymin;
	block_total[threadIdx.x] = mytotal;

	__syncthreads();

//	if(threadIdx.x == 0) {
//		mymax = INT_MIN;
//		mymin = INT_MAX;
//		mytotal = 0;
//		for(int i = 0; i < blockDim.x; i++) {
//			mytotal += block_total[i];
//			mymax = block_max[i] > mymax ? block_max[i] : mymax;
//			mymin = block_min[i] < mymin ? block_min[i] : mymin;
//		}
//
//		device_max[blockIdx.x] = mymax;
//		device_min[blockIdx.x] = mymin;
//		device_total[blockIdx.x] = mytotal;
//	}

//	mytotal = block_total[threadIdx.x];
//	mymax = block_max[threadIdx.x];
//    mymin = block_min[threadIdx.x];

	int half_block = blockDim.x >> 1;
	for (int half = half_block; half > 0; half>>=1) {//log

		if(threadIdx.x < half){

			mytotal = block_total[threadIdx.x];
			mymax = block_max[threadIdx.x];
			mymin = block_min[threadIdx.x];

			valueNeighbor = block_max[threadIdx.x + half];
			if(mymax < valueNeighbor){
				mymax = valueNeighbor;
				block_max[threadIdx.x] = mymax;
			}

			valueNeighbor = block_min[threadIdx.x + half];
			if(mymin > valueNeighbor){
				mymin = valueNeighbor;
				block_min[threadIdx.x] = mymin;
			}

			valueNeighbor = block_total[threadIdx.x + half];
			mytotal += valueNeighbor;
			block_total[threadIdx.x] = mytotal;
		}

		__syncthreads();
	}

	if(idWarp == 0 && idtW == 0)
		device_max[blockIdx.x] = block_max[0];

	if(idWarp == 1 && idtW == 0)
		device_min[blockIdx.x] = block_min[0];

	if(idWarp == 2 && idtW == 0)
		device_total[blockIdx.x] = block_total[0];

//	#pragma unroll 4
//	for (int i = 16; i > 0; i>>=1) { //log
//		valueNeighbor = __shfl_down(mymax, i);
//		if(mymax < valueNeighbor){
//			mymax = valueNeighbor;
//		}
//
//		valueNeighbor = __shfl_down(mymin, i);
//		if(mymin > valueNeighbor){
//			mymin = valueNeighbor;
//		}
//
//		valueNeighbor = __shfl_down(mytotal, i);
//		mytotal += valueNeighbor;
//	}
//
//	if(idtW == 0){
//		block_max[idWarp] = mymax;
//		block_min[idWarp] = mymin;
//		block_total[idWarp] = mytotal;
//	}
//
//	__syncthreads();

//	if(idWarp == 0){
//		mytotal = block_total[idtW];
//		mymax = block_max[idtW];
//	    mymin = block_min[idtW];
//
//		#pragma unroll 4
//		for (int i = 16; i > 0; i>>=1) { //log
//			valueNeighbor = __shfl_down(mymax, i);
//
//			if(mymax < valueNeighbor){
//				mymax = valueNeighbor;
//			}
//
//			valueNeighbor = __shfl_down(mymin, i);
//			if(mymin > valueNeighbor){
//				mymin = valueNeighbor;
//			}
//
//			valueNeighbor = __shfl_down(mytotal, i);
//			mytotal += valueNeighbor;
//		}
//
//		if(idtW == 0){
//			block_max[idWarp] = mymax;
//			block_min[idWarp] = mymin;
//			block_total[idWarp] = mytotal;
//		}
//	}
//
//	__syncthreads();
//
//	if(idWarp == 0 && idtW == 0)
//		device_max[blockIdx.x] = mymax;
//
//	if(idWarp == 1 && idtW == 0)
//		device_min[blockIdx.x] = block_min[0];
//
//	if(idWarp == 2 && idtW == 0)
//		device_total[blockIdx.x] = block_total[0];
}

int main() {
	
	/*Leitura da entrada*/
	int n;
	scanf("%d",&n);	

	int * input = (int *)malloc(n*sizeof(int));

	//only to check
	int * psum = (int *)malloc(n*sizeof(int));
	int * ssum = (int *)malloc(n*sizeof(int));
	int * pmax = (int *)malloc(n*sizeof(int));
	int * smax = (int *)malloc(n*sizeof(int));
	int * m    = (int *)malloc(n*sizeof(int));
	int * segmentedscan    = (int *)malloc(n*sizeof(int));

	for(int i = 0; i < n; i++) {
		scanf("%d",&input[i]);
	}		

	int devCount;
	HANDLE_ERROR( hipGetDeviceCount(&devCount));

	thrust::host_vector<int> spawn_right(devCount);
	thrust::host_vector<int> spawn_left(devCount);

	thrust::host_vector<int> local_max(devCount);
	thrust::host_vector<int> local_min(devCount);
	thrust::host_vector<int> local_total(devCount);	

	#pragma omp parallel num_threads(devCount) default(shared)
	{
		hipEvent_t start,stop,stop2;
		float time1, time2;
		const int dev_id = omp_get_thread_num();

		HANDLE_ERROR( hipSetDevice(dev_id) );

		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, dev_id);


//		for(int j=0;j<devCount;j++) {
//			if(dev_id != j) {
//				int access;
//				hipDeviceCanAccessPeer(&access,dev_id,j);
//				if(access) {
//					hipDeviceEnablePeerAccess(j,0);
//				}
//			}
//		}

		int partition = (int)(n/devCount);
		if(dev_id == devCount -1) {
			if(n % devCount != 0)
				partition += n % devCount;
		}

		thrust::device_vector<int> d_q(partition);
		thrust::device_vector<int> d_psum(partition);
		thrust::device_vector<int> d_ssum(partition);
		thrust::device_vector<int> d_smax(partition);
		thrust::device_vector<int> d_pmax(partition);
		thrust::device_vector<int> d_m(partition);

		thrust::device_vector<int> d_segmentedscan(partition);

		int num_threads = THREAD_NUM;
		int num_blocks = devProp.multiProcessorCount*BLOCKS_SCHED;

		dim3 threadsPorBloco(num_threads);
		dim3 blocosPorGrid(num_blocks);

		thrust::device_vector<int> d_result_max(num_blocks);
		thrust::device_vector<int> d_result_min(num_blocks);
		thrust::device_vector<int> d_result_total(num_blocks);

		thrust::device_vector<int>::iterator iter;

		thrust::fill(thrust::device, d_segmentedscan.begin(), d_segmentedscan.end(), 1);
		thrust::fill(thrust::device, d_result_max.begin(), d_result_max.end(), INT_MIN);
		thrust::fill(thrust::device, d_result_min.begin(), d_result_min.end(), INT_MAX);
		thrust::fill(thrust::device, d_result_total.begin(), d_result_total.end(), 0);

		thrust::copy(input + partition*dev_id, input + partition*dev_id + partition, d_q.begin());

		int correct1, correct2 = 0;
		int k, l;
		int global_max;		

		/*Tomada de tempo na GPU-0*/
		if(dev_id == 0) {
			HANDLE_ERROR( hipEventCreate(&start) );
			HANDLE_ERROR( hipEventCreate(&stop) );
			HANDLE_ERROR( hipEventCreate(&stop2) );
			HANDLE_ERROR( hipEventRecord(start, 0) );
		}

		thrust::inclusive_scan(d_q.begin(), d_q.end(), d_psum.begin());
		thrust::inclusive_scan(d_q.rbegin(), d_q.rend(), d_ssum.rbegin());

		/*IMPROVE AllToAll - Now it's using host memory*/
		spawn_right[dev_id] = d_psum[partition - 1];
		spawn_left[dev_id] = d_ssum[0];

		HANDLE_ERROR( hipDeviceSynchronize() );
		#pragma omp barrier

		correct1 = 0; correct2 = 0;
		k = 0, l = devCount - 1;
		while(k < dev_id || l > dev_id) {
			if(k < dev_id)
				correct1 += spawn_right[k];

			if(l > dev_id)
				correct2 += spawn_left[l];

			k++; l--;
		}

		if(dev_id != 0)
			thrust::for_each(d_psum.begin(), d_psum.end(), _1 += correct1);

		if(dev_id != devCount - 1)
			thrust::for_each(d_ssum.rbegin(), d_ssum.rend(), _1 += correct2);
			

		thrust::inclusive_scan(d_psum.rbegin(), d_psum.rend(), d_smax.rbegin(), max_op);
		thrust::inclusive_scan(d_ssum.begin(), d_ssum.end(), d_pmax.begin(), max_op);

		/*IMPROVE AllToAll -Now it's using host memory*/
		spawn_right[dev_id] = d_pmax[partition - 1];
		spawn_left[dev_id] = d_smax[0];

		HANDLE_ERROR( hipDeviceSynchronize() );
		#pragma omp barrier

		correct1 = INT_MIN; correct2 = INT_MIN;
		k = 0, l = devCount - 1;
		while(k < dev_id || l > dev_id) {
			if(k < dev_id)
				correct1 = spawn_right[k] >  correct1 ? spawn_right[k] : correct1;

			if(l > dev_id)
				correct2 = spawn_left[l] > correct2 ? spawn_left[l] : correct2;

			k++; l--;
		}

		if(dev_id != 0) {
			//Optimize it!!
			if(correct1 > d_pmax[0]) {
				d_pmax[0] = correct1;
				thrust::inclusive_scan(d_pmax.begin(), d_pmax.end(), d_pmax.begin(), max_op);
			}
		}

		if(dev_id != devCount - 1) {
			//Optimize it!!
			if(correct2 > d_smax[partition - 1]) {
				d_smax[partition - 1] = correct2;
				thrust::inclusive_scan(d_smax.rbegin(), d_smax.rend(), d_smax.rbegin(), max_op);
			}			
		}

		/*Cálculo do vetor M*/
	        thrust::transform(thrust::device, 
			thrust::make_zip_iterator(make_tuple(d_q.begin(),d_psum.begin(),d_ssum.begin(),d_pmax.begin(),d_smax.begin())),
			thrust::make_zip_iterator(make_tuple(d_q.end(),d_psum.end(),d_ssum.end(),d_pmax.end(),d_smax.end())),		
			       d_m.begin(),
			       calcM() );

		local_max[dev_id] = thrust::reduce(thrust::device, d_m.begin(), d_m.end(), INT_MIN, max_op);

		#pragma omp barrier

		global_max = thrust::reduce(thrust::host, local_max.begin(), local_max.end(), INT_MIN, max_op);

		/*Finalizando a tomada de tempo - Primeira parte - Encontrar o máximo*/
		if(dev_id == 0) {
			HANDLE_ERROR( hipEventRecord(stop, 0) );
			HANDLE_ERROR( hipEventSynchronize(stop) );
			HANDLE_ERROR( hipEventElapsedTime(&time1, start, stop) );
		}

		thrust::for_each(d_m.begin(), d_m.end(), binaryTransform(global_max));

		thrust::inclusive_scan_by_key(thrust::device, d_m.begin(), d_m.end(), d_segmentedscan.begin(), d_segmentedscan.begin());

		thrust::transform(thrust::device, d_m.begin(), d_m.end(), d_segmentedscan.begin(), d_segmentedscan.begin(), bit_and_op);

		/*IMPROVE AllToAll -Now it's using host memory*/
		spawn_right[dev_id] = d_segmentedscan[partition - 1];

		HANDLE_ERROR( hipDeviceSynchronize() );
		#pragma omp barrier

		correct1 = 0;
		l = dev_id - 1;
		while(l >= 0) {
			if(spawn_right[l] > 0) {
				correct1 += spawn_right[l];
			}
			if(spawn_right[l] == partition)
				l--;
			else
				break;
		}

		iter = thrust::find(thrust::device, d_segmentedscan.begin(), d_segmentedscan.end(), 0);
		thrust::for_each(d_segmentedscan.begin(), iter, _1 += correct1);

		int * segscan = thrust::raw_pointer_cast(d_segmentedscan.data());
		int * dmax = thrust::raw_pointer_cast(d_result_max.data());
		int * dmin = thrust::raw_pointer_cast(d_result_min.data());
		int * dtotal = thrust::raw_pointer_cast(d_result_total.data());

		/*IMPROVE AllToAll -Now it's using host memory*/
		spawn_left[dev_id] = d_segmentedscan[0];

		HANDLE_ERROR( hipDeviceSynchronize() );
		#pragma omp barrier

		int boundary = dev_id < devCount - 1 ? spawn_left[dev_id+1] : 0;
		findRelatedSolutions<<<blocosPorGrid,threadsPorBloco, 3*num_threads*sizeof(int)>>>(segscan,partition, dev_id, boundary, dmax, dmin, dtotal);
		CudaCheckError();

		local_max[dev_id] = thrust::reduce(thrust::device, d_result_max.begin(), d_result_max.end(), INT_MIN, max_op);
		local_min[dev_id] = thrust::reduce(thrust::device, d_result_min.begin(), d_result_min.end(), INT_MAX, min_op);
		local_total[dev_id] = thrust::reduce(thrust::device, d_result_total.begin(), d_result_total.end(), 0, plus_op);

		HANDLE_ERROR( hipDeviceSynchronize() );
		#pragma omp barrier

		int maximum = INT_MIN;
		int minimum = INT_MAX;
		int total = 0;
		for(int i = 0; i < devCount; i++) {
			maximum = local_max[i] > maximum ? local_max[i] : maximum;
			minimum = local_min[i] < minimum ? local_min[i] : minimum;
			total += local_total[i];
		}
		

		/*Finalizando a tomada de tempo - Segunda parte - Problemas Relacionados*/
		if(dev_id == 0) {
			HANDLE_ERROR( hipEventRecord(stop2, 0) );
			HANDLE_ERROR( hipEventSynchronize(stop2) );
			HANDLE_ERROR( hipEventElapsedTime(&time2, stop, stop2) );
			printf("%.9f %.9f = mss=%d max=%d min=%d total=%d\n",time1,time2,global_max,maximum,minimum,total);
		}

		thrust::copy(d_psum.begin(), d_psum.end(), psum + partition*dev_id);
		thrust::copy(d_ssum.begin(), d_ssum.end(), ssum + partition*dev_id);
		thrust::copy(d_pmax.begin(), d_pmax.end(), pmax + partition*dev_id);
		thrust::copy(d_smax.begin(), d_smax.end(), smax + partition*dev_id);
		thrust::copy(d_m.begin(), d_m.end(), m + partition*dev_id);
		thrust::copy(d_segmentedscan.begin(), d_segmentedscan.end(), segmentedscan + partition*dev_id);

		#pragma omp barrier	

	}

	/*printf("input: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",input[i]);
	}
	printf("\n");

	printf("psum: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",psum[i]);
	}
	printf("\n");

	printf("ssum: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",ssum[i]);
	}
	printf("\n");

	printf("pmax: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",pmax[i]);
	}
	printf("\n");

	printf("smax: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",smax[i]);
	}
	printf("\n");

	printf("m: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",m[i]);
	}
	printf("\n");

	printf("local_max: \n");
	for(int i = 0; i < devCount; i++) {
		printf("%d ",local_max[i]);
	}
	printf("\n");

	printf("local_min: \n");
	for(int i = 0; i < devCount; i++) {
		printf("%d ",local_min[i]);
	}
	printf("\n");

	printf("local_total: \n");
	for(int i = 0; i < devCount; i++) {
		printf("%d ",local_total[i]);
	}
	printf("\n");

	printf("seg-scan: \n");
	for(int i = 0; i < n; i++) {
		printf("%d ",segmentedscan[i]);
	}
	printf("\n");

	printf("boundary: \n");
	for(int i = 0; i < devCount; i++) {
		printf("%d ",spawn_left[i]);
	}
	printf("\n");*/



	free(input);

	free(psum);
	free(ssum);
	free(pmax);
	free(smax);
	free(m);
	free(segmentedscan);

	return 0;
}
